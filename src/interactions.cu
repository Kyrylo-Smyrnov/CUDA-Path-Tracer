#include "hip/hip_runtime.h"
#include "interactions.h"

__host__ __device__ glm::vec3 calculateRandomDirectionInHemisphere(
    glm::vec3 normal,
    thrust::default_random_engine &rng)
{
    thrust::uniform_real_distribution<float> u01(0, 1);

    float up = sqrt(u01(rng)); // cos(theta)
    float over = sqrt(1 - up * up); // sin(theta)
    float around = u01(rng) * TWO_PI;

    // Find a direction that is not the normal based off of whether or not the
    // normal's components are all equal to sqrt(1/3) or whether or not at
    // least one component is less than sqrt(1/3). Learned this trick from
    // Peter Kutz.

    glm::vec3 directionNotNormal;
    if (abs(normal.x) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(1, 0, 0);
    }
    else if (abs(normal.y) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(0, 1, 0);
    }
    else
    {
        directionNotNormal = glm::vec3(0, 0, 1);
    }

    // Use not-normal direction to generate two perpendicular directions
    glm::vec3 perpendicularDirection1 =
        glm::normalize(glm::cross(normal, directionNotNormal));
    glm::vec3 perpendicularDirection2 =
        glm::normalize(glm::cross(normal, perpendicularDirection1));

    return up * normal
        + cos(around) * over * perpendicularDirection1
        + sin(around) * over * perpendicularDirection2;
}

__host__ __device__ glm::vec3 calculateRandomSpecularDirection(
    glm::vec3 R,
    float exponent,
    thrust::default_random_engine& rng
)
{
    thrust::uniform_real_distribution<float> u01(0, 1);

    float theta = glm::acos(pow((double)(u01(rng)), (double)(1 / (exponent + 1))));
    float phi = TWO_PI * u01(rng);

    float x = cos(phi) * sin(theta);
    float y = sin(phi) * sin(theta);
    float z = cos(theta);

    glm::vec3 directionNotR;
    if (abs(R.x) < SQRT_OF_ONE_THIRD)
    {
        directionNotR = glm::vec3(1, 0, 0);
    }
    else if (abs(R.y) < SQRT_OF_ONE_THIRD)
    {
        directionNotR = glm::vec3(0, 1, 0);
    }
    else
    {
        directionNotR = glm::vec3(0, 0, 1);
    }

    glm::vec3 tangent = glm::normalize(glm::cross(R, directionNotR));
    glm::vec3 bitangent = glm::normalize(glm::cross(R, tangent));

    return x * tangent + y * bitangent + z * R;
}

__host__ __device__ void scatterRay(
    PathSegment & pathSegment,
    glm::vec3 intersect,
    glm::vec3 normal,
    const Material &m,
    thrust::default_random_engine &rng)
{
    if (m.hasReflective > 0.0f)
    {
        if (m.specular.exponent == INFINITY)
        {
            pathSegment.ray.direction = glm::reflect(pathSegment.ray.direction, normal);
            pathSegment.ray.origin = intersect + pathSegment.ray.direction * EPSILON;
        }
        else
        {
            glm::vec3 R = glm::reflect(pathSegment.ray.direction, normal);

            pathSegment.ray.direction = glm::normalize(calculateRandomSpecularDirection(R, m.specular.exponent, rng));
            pathSegment.ray.origin = intersect + pathSegment.ray.direction * EPSILON;
        }
    }
    else {
        pathSegment.ray.direction = glm::normalize(calculateRandomDirectionInHemisphere(normal, rng));
        pathSegment.ray.origin = intersect + normal * EPSILON;
    }

    pathSegment.remainingBounces--;
}
